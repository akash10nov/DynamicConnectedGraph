
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
#include <stdint.h>
#define MAX 100
#define MIN 1
__global__ void DUKernel(int *D_Level,int *D_Del,int n, int num);
void IscomponentSame(int *L,int *D, int n,int num);

uint64_t getTime(){
	struct timeval t;
	gettimeofday(&t, NULL);
	return (uint64_t)(t.tv_sec)*1000000 + (uint64_t)(t.tv_usec);
}
int ModifyMatrix_node_seperation(int* Mat,int n, int index)
{
	/* for node seperation
	*  delete one whole row
	*  and respective edges
	*  from other rows.
	*/
	int i=0;
	//int j=0;
	
	for(i=0;i<n;i++)
	{
		Mat[(i*n)+index]=0;
		Mat[(index*n)+i]=0;
	}

	return 1;
}



int ModifyMatrix_random_deletion(int* Mat,int n, int Num,int *del_list)
{
	int i;
	int j;
	int count=0;
	i=0;
	j=0;
	int k=0;
	for(k=0;k<n;k++)
	{
		i=(rand()%n);
		for(j=0;j<n;j++)
		{
			if(Mat[(i*n)+j]==1)
			{
				Mat[(i*n)+j]=0;
				Mat[(j*n)+i]=Mat[(i*n)+j];
				del_list[(count*2)+0]=i;
				del_list[(count*2)+1]=j;
				count++;
				if(count==Num)
					return 1;
				break;
			}
		}
	}


	return 1;

}


int ModifyMatrix_insertion(int* Mat,int n)
{

	int i;
	int j;
	//int flag=0;
	j=0;
	int k=0;
	for(k=0;k<n;k++)
	{
		i=(rand()%n);
		for(j=0;j<n;j++)
		{
			if(Mat[(i*n)+j]==0)
			{
				Mat[(i*n)+j]=1;
				Mat[(j*n)+i]=Mat[(i*n)+j];
				return 1;
			}
		}
	}


	return 1;


}


int ConnectedGraph_checker(int* Mat,int* elements_covered,int* queue,int n)
{
			//printf("CHECK.1\n");

			elements_covered[0]=1;
			int add=0;
			//printf("CHECK.2\n");

			for(int j=1;j<n;j++)
			{
				if(Mat[j]==1)
				{
					elements_covered[j]=2;
					//printf("q:index:%d  value:%d\n",add,j);
					queue[add++]=j;
				}
			}
			//printf("CHECK.3\n");

			for(int i=1;i<n;i++)
			{
				for(int j=0;j<n;j++)
				{
					//printf("%2d %2d\n",i,j);
				
					if(queue[i]>-1 && Mat[(queue[i]*n)+j]==1 && i!=j && elements_covered[j]<1 )
					{
						//printf("q:add:%d  value:%d\n",add,j);
						queue[add++]=j;
						elements_covered[j]=elements_covered[queue[i]]+1;
					}
				}
			}
		//printf("CHECK.4\n");

		for(int i=0;i<n;i++)
		{
			if(elements_covered[i]<0)
			{
				printf("%d\n",i);
				return -1;
			}
		}
		//printf("CHECK.5\n");

		return 1;
	
	
}

int main(int argc, char **argv)
{
	int n;
	//printf("Num of vertices:\n");
	//scanf("%d",&n);
	n=atoi(argv[1]);
	int index;
	
	/* declare an 2-D array to store information about edges*/
	int* adj_matrix;
	adj_matrix=(int*) malloc ((sizeof(int*))*n*n);
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=i;j<n;j++)
		{
			int p;
			p=(rand() % n);
			int r;
			r=p/(n-15);
			//printf("%d\n",r);
			adj_matrix[(i*n)+j]=r;
			adj_matrix[(j*n)+i]=adj_matrix[(i*n)+j];
		}
	}
	int count=0;
	for(i=0;i<n;i++)
	{
		for(j=i;j<n;j++)
		{
			if(adj_matrix[(i*n)+j]==1)
				count++;
		}
		//printf("\n");
	}
	//printf("No of vertices: %d\n",n);
	count=0;

	int* elements_covered;
	elements_covered=(int*) malloc(sizeof(int)*n);
	int* queue;
	queue=(int*) malloc(sizeof(int)*n);
	for(i=0;i<n;i++)
	{
		elements_covered[i]=-1;
	}
	/*check the whether graph is connected or not*/
	int true_false=ConnectedGraph_checker(adj_matrix,elements_covered,queue,n);
	
	if(true_false==1)
	{
		//printf("Graph is connected\n");
	}
	else
		printf("Graph is NOT connected\n");
	for(i=0;i<n;i++)
	{
		//elements_covered[i]=-1;
		queue[i]=-1;
	}
	//printf("1\n");
	//struct timeval start, end;
	//printf("2\n");

	//int p;
	//gettimeofday(&start, NULL);
	//printf("3\n");
	//printf("Modify type:\n");
	//printf("1.detach one node\n");
	//printf("2.Random deletion\n");
	//printf("3.insertion.\n");
	int type;
	//scanf("%d",&type);
	type=2;
	int Num;
	int *deletions;
	if(type==1)
	{
		printf("Node index to be removed:\n");
		scanf("%d",&index);
	}
	if(type==2)
	{
		//printf("How many edge deletion:\n");
		Num=atoi(argv[2]);
		//scanf("%d",&Num);
		deletions=(int *)malloc(sizeof(int)*2*Num);
	}
	int p=0;
	//printf("Edges removes:%d:\n",Num);

	if(type==1)
		p=ModifyMatrix_node_seperation(adj_matrix,n,index);
	if(type==2)
		p=ModifyMatrix_random_deletion(adj_matrix,n,Num,deletions);
	if(type==3)
		p=ModifyMatrix_insertion(adj_matrix,n);
	//printf("4\n");
	
	/*
	for(i=0;i<Num;i++)
	{
		//printf("edge to be deleted:%d %2d\n",deletions[(i*2)],deletions[(i*2)+1]);
		//printf("their weight:%d %2d\n",elements_covered[deletions[(i*2)]],elements_covered[deletions[(i*2)+1]]);

		
	}

	*/
	
	//int flag=1;
	
	IscomponentSame(elements_covered,deletions,n,Num);
	/*
		for(int i=0;i<n;i++)
		printf("ele:%d \n",elements_covered[i]);
	*/
	int* harmful;
	harmful=(int*)malloc((sizeof(int)*Num));
	for(i=0;i<Num;i++)
	{
		//printf("edge:%d. %2d %2d\n",i,deletions[(i*2)],deletions[(i*2)+1]);
		if(deletions[(i*2)]<-1)
		{
			int Vertice=(deletions[(i*2)]+2)/(-2);
			for(j=0;j<n;j++)
			{
				if(j!=Vertice && adj_matrix[(Vertice*n)+j]==1)
				{
					if(elements_covered[Vertice]>=elements_covered[j])
					{
						//printf("num %d. Deletion was safe.\n",i);
						harmful[i]=1;
						break;
					}
				}	
			}
		}
			
		else if(deletions[(i*2)+1]<-1)
		{
			int Vertice=(deletions[(i*2)]+2)/(-2);
			for(j=0;j<n;j++)
			{
				if(j!=Vertice && adj_matrix[(Vertice*n)+j]==1)
				{
					if(elements_covered[Vertice]>=elements_covered[j])
					{
						//printf("num %d. Deletion was safe.\n",i);
						harmful[i]=1;

						break;
					}
				}	
			}
		}
		else if(deletions[(i*2)]==-1)
		{
			//printf("num %d. Deletion was safe.\n",i);
			harmful[i]=1;
		}
					
	}
	int s=0;
	for(i=0;i<Num;i++)
	{
		if(harmful[i]!=1)
			s=1;//printf("%d deletion would divide graph into 2 parts\n",i);
		else
		{
		}
	}


	return 0;
}
void IscomponentSame(int *L,int *D, int n,int num) 
{ 
    int *d_Level ,*d_Deletions; 
    uint64_t astart, aend;
   
    hipMalloc(&d_Level, n*sizeof(int));
    hipMalloc(&d_Deletions, num*2*sizeof(int));
   int i=0;
    //Copying data to device from host 
   astart = getTime();
    hipMemcpy(d_Level, L, sizeof(int)*n,hipMemcpyHostToDevice);
    hipMemcpy(d_Deletions, D, sizeof(int)*(num*2),hipMemcpyHostToDevice);
	
     
   DUKernel<<<1 ,num,4*num*2>>>(d_Level, d_Deletions ,n,num); 
   
   //cudaMemcpy(L, d_Level, sizeof(int)*(n),cudaMemcpyDeviceToHost);
   
   hipMemcpy(D, d_Deletions, sizeof(int)*(num*2),hipMemcpyDeviceToHost);
	 aend = getTime();
    //Deallocating memory on the device 
    hipFree(d_Level); 
    hipFree(d_Deletions);
   
	
    printf("%f,%10d,%10d \n",(aend-astart)/1000000.0,n,num); 
}


__global__ void DUKernel(int *D_Level,int *D_Del,int n, int num)
{
	// 10x10 size matrix is for experiment, so argv[1]=10
 
   	 extern __shared__ int temp[];	 
	   int p=threadIdx.x;
	 //int j=blockIdx.x;
	 //int p= threadIdx.x+(blockIdx.x*blockDim.x);
	 temp[p*2]= D_Del[(p*2)];
	 temp[(p*2)+1]= D_Del[(p*2)+1];

	 __syncthreads();
	//int i=0;
	
	/*int s=0;
	 while(i<threadIdx.x && s< blockIdx.x)
	 {
		temp[p]=temp[p]-(temp[(s*dimension)+(k*(j/1000))+k] * ((temp[(j*dimension)+(i*(j/1000))+i])/temp[(j*dimension)+(j*(j/1000))+j]));
		i++;
		s++;
	 }
	 */
	//printf("threadID:%d\n",p);
	//printf("out: %d %2d  %d\n",D_Level[D_Del[(p*2)]],D_Level[D_Del[(p*2)+1]],p);
	 if(D_Level[D_Del[(p*2)]]> D_Level[D_Del[(p*2)+1]])
	 {
		 temp[(p*2)]=(temp[(p*2)]*-2)-2;
		// printf("Kernel: %d %2d %2d\n",D_Del[(p*2)],D_Del[(p*2)+1], p);

	 }
	 else if(D_Level[D_Del[(p*2)]]< D_Level[D_Del[(p*2)+1]])
	 {
		temp[(p*2)+1]=(temp[(p*2)+1]*-2)-2;
		 //printf("Kernel: %d %2d %2d\n",D_Del[(p*2)],D_Del[(p*2)+1], p);
	 }
	 else
	 {
		 temp[(p*2)]=-1;
		 temp[(p*2)+1]=-1;
		 //printf("Kernel: %d %2d %2d\n",D_Del[(p*2)],D_Del[(p*2)+1], p);

	 }
	//__syncthreads();
	D_Del[(p*2)]=temp[(p*2)+1];
	D_Del[(p*2)]=temp[(p*2)];


}
